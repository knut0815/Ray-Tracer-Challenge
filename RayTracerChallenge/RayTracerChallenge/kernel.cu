#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Canvas.h"
#include "Sphere.h"
#include "Material.h"
#include "PointLight.h"
#include "Ray.h"
#include "Matrix4.h"
#include "Vec4.h"
#include "Camera.h"
#include "World.h"
#include "Colors.h"


#define THREADS_PER_BLOCK 256


__global__ void DrawScene(unsigned int *colorsCuda, World *world, Camera *camera, int width, int height) //int *checkerCuda, 
{
	int val = threadIdx.x + blockIdx.x * blockDim.x;
	int i = val % width;
	int j = height - 1 - val / width;


	Ray ray = camera->RayAtPixel(i, j);
	
	Colorf color = world->ColorAt(ray);

	int r = (int)(color.r * 255.999f);
	int g = (int)(color.g * 255.999f);
	int b = (int)(color.b * 255.999f);

	r = r > 255 ? 255 : r;
	g = g > 255 ? 255 : g;
	b = b > 255 ? 255 : b;

	unsigned int dword;

	dword = (((unsigned char)r << 16u) | ((unsigned char)g << 8u) | (unsigned char)b);

	colorsCuda[val] = dword;
}

int mainCUDA(unsigned int *colors, Camera &camera)
{

	World world;


	//int *checkerCuda;
	unsigned int *colorsCuda;
	World *worldCuda;
	Camera *cameraCuda;

	//hipMalloc((void**)&checkerCuda, camera.width * camera.height * sizeof(int));
	hipMalloc((void**)&colorsCuda, camera.width * camera.height * sizeof(unsigned int));
	hipMalloc((void**)&worldCuda, sizeof(World));
	hipMalloc((void**)&cameraCuda, sizeof(Camera));

	//hipMemcpy(checkerCuda, checker, camera.width * camera.height * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(colorsCuda, colors, camera.width * camera.height * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(worldCuda, &world, sizeof(World), hipMemcpyHostToDevice);
	hipMemcpy(cameraCuda, &camera, sizeof(Camera), hipMemcpyHostToDevice);


	int blockSize = THREADS_PER_BLOCK;
	int numBlocks = (camera.width * camera.height + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;

	DrawScene <<< numBlocks, blockSize >>> (colorsCuda, worldCuda, cameraCuda, camera.width, camera.height); //checkerCuda, 
	hipDeviceSynchronize();

	//hipMemcpy(checker, checkerCuda, camera.width * camera.height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(colors, colorsCuda, camera.width * camera.height * sizeof(unsigned int), hipMemcpyDeviceToHost);


	//hipFree(checkerCuda);
	hipFree(colorsCuda);
	hipFree(worldCuda);
	hipFree(cameraCuda);


	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


    return 0;
}